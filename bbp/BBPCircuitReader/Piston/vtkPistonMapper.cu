#include "hip/hip_runtime.h"
#ifdef _WIN32
#include <windows.h>
#endif

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/functional.h>
#include <thrust/sort.h>

#include <cuda_gl_interop.h>

#include "vtkScalarsToColors.h"
#include "vtkPistonDataObject.h"
#include "vtkPistonDataWrangling.h"
#include "vtkPistonScalarsColors.h"
#include "vtkPistonMinMax.h"
#include "vtkPistonReference.h"
#include "piston/piston_math.h"

#include "vtkgl.h"

#include <iostream>

using namespace std;

namespace vtkpiston {

bool AlmostEqualRelativeAndAbs(float A, float B,
            float maxDiff, float maxRelDiff)
  {
    // Check if the numbers are really close -- needed
    // when comparing numbers near zero.
    float diff = fabs(A - B);
    if (diff <= maxDiff)
        return true;

    A = fabs(A);
    B = fabs(B);
    float largest = (B > A) ? B : A;

    if (diff <= largest * maxRelDiff)
        return true;
    return false;
  }

template <typename ValueType>
struct color_map : thrust::unary_function<ValueType, float3>
{
    const ValueType min;
    const ValueType max;
    const int size;
    float *table;
    const int numberOfChanels;

    color_map(float *table, int arrSize, int noOfChanels,
      ValueType rMin, ValueType rMax) :
      min(rMin),
      max(rMax),
      size((arrSize / noOfChanels) - 1),
      table(table),
      numberOfChanels(noOfChanels)
      {
      }

    __host__ __device__
    float3 operator()(ValueType val)
    {
      int index = 0;
      if((max - min) > 0.0)
        {
        index = ( (val - min) / (max - min) ) * size;
        }

      if (index < 0) index = 0;
      if (index > size) index = size;
      index *= numberOfChanels;

      float3 color;
      if(numberOfChanels == 1)
        {
        color = make_float3(table[index], table[index], table[index]);
        }
      else if(numberOfChanels == 2)
        {
        color = make_float3(table[index], table[index + 1], 0.0f);
        }
      else if(numberOfChanels == 3)
        {
        color = make_float3(table[index], table[index + 1], table[index + 2]);
        }
      else
        {
        // Not supported
        }

      return color;
    }
};

//------------------------------------------------------------------------------
void CudaGLInit()
{
  hipDeviceProp_t prop;
  int dev;

  // Fill it with zeros
  memset(&prop,0,sizeof(hipDeviceProp_t));

  // Pick a GPU capable of 1.0 or better
  prop.major=1; prop.minor=0;
  hipChooseDevice(&dev,&prop);

  // Set OpenGL device
  hipError_t res = cudaGLSetGLDevice(dev);

  if (res != hipSuccess)
    {
    cerr << "Set device failed  ... " << hipGetErrorString(res) << endl;
    return;
    }
}

//------------------------------------------------------------------------------
void CudaRegisterBuffer(struct hipGraphicsResource **vboResource,
                        GLuint vboBuffer)
{
  hipError_t res =
    hipGraphicsGLRegisterBuffer(vboResource, vboBuffer,
                                cudaGraphicsMapFlagsWriteDiscard);
  if (res != hipSuccess)
  {
    cerr << "Register buffer failed ... " << hipGetErrorString(res) << endl;
    return;
  }
}


//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
struct distance_functor 
{
  float3 cameravector;

  // construct with a constant camera vector
  __host__ __device__ distance_functor(float3 &cam) : cameravector(cam) {}

  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  {
    thrust::get<1>(t) = dot(thrust::get<0>(t), cameravector);  
  }
};
//------------------------------------------------------------------------------
struct celldistance_functor 
{
  const float *vertex_distances;
  
  // construct with a precomputed distance vector for every vertex
  __host__ __device__ celldistance_functor(float *v) : vertex_distances(v) {}
  
  template <typename Tuple>
  __host__ __device__
  void operator()(Tuple t)
  { 
    thrust::get<1>(t) = (vertex_distances[thrust::get<0>(t).x] + 
                         vertex_distances[thrust::get<0>(t).y] +
                         vertex_distances[thrust::get<0>(t).z])/3.0;
  }
};
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
//------------------------------------------------------------------------------
void DepthSortPolygons(vtkPistonDataObject *id, double *cameravec)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL) {
    // Type mismatch, don't bother trying
    return;
  }
  vtk_polydata *pD = (vtk_polydata *)tr->data;

  //
  // we need to compute the distance to the camera for each cell.
  // Perform a dot product of each vertex with the supplied camera vector
  //

  // prepare an array for the distances
  thrust::device_vector<float> distances(pD->points->size());

  // initialize our functor which will compute distance and store in a vector
  float3 cam = make_float3(cameravec[0], cameravec[1], cameravec[2]);
  distance_functor distance(cam);

  // apply distance functor using input and output arrays using zip_iterator
  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->begin(), distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->points->end(),   distances.end())),
    distance);

  // to test if it is working, copy the distances into the scalars
  // so we can colour by scalar values
//  thrust::copy(distances.begin(), distances.end(), pD->scalars->begin());
  
  //
  // To compute the average distance for each cell, we must
  // sum/gather 3 distances (one for each vertex) for every cell by
  // looking up the vertex indices from the cell array tuples
  //

  // prepare an array for the distances
  thrust::device_vector<float> cell_distances(pD->nCells);

  celldistance_functor celldist(thrust::raw_pointer_cast(distances.data()));

  thrust::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->begin(), cell_distances.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(pD->cells->end(),   cell_distances.end())),
    celldist);

  //
  // now we want to sort the cells using the average distance
  // we must copy the cell vertex index tuple during the sort
  //
  thrust::sort_by_key(cell_distances.begin(), cell_distances.end(), pD->cells->begin(), 
    thrust::greater<float>());
}

//------------------------------------------------------------------------------
void CudaTransferToGL(vtkPistonDataObject *id, unsigned long dataObjectMTimeCache,
                      vtkPistonScalarsColors *psc,
                      hipGraphicsResource **vboResources,
                      bool &hasNormals, bool &hasColors, 
                      bool &useindexbuffers)
{
  vtkPistonReference *tr = id->GetReference();
  if (tr->type != VTK_POLY_DATA || tr->data == NULL)
    {
    // Type mismatch, don't bother trying
    return;
    }

  vtk_polydata *pD = (vtk_polydata *)tr->data;

  // Claim access to buffer for cuda
  hipError_t res;
  res = hipGraphicsMapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Claim for CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  size_t num_bytes;
  float3 *vertexBufferData;
  uint3  *cellsBufferData;
  float  *normalsBufferData;
  // float3 *colorsBufferData;
  float4  *colorbufferdata; 

  res = hipGraphicsResourceGetMappedPointer
      ((void **)&vertexBufferData, &num_bytes, vboResources[0]);
  if(res != hipSuccess) {
    cerr << "Get mappedpointer for vertices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&normalsBufferData, &num_bytes, vboResources[1]);
  if(res != hipSuccess) {
    cerr << "Get mappedpointer for normals failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }
  res = hipGraphicsResourceGetMappedPointer
      ((void **)&colorbufferdata, &num_bytes, vboResources[2]);
  if(res != hipSuccess)
  {
    cerr << "Get mappedpointer for colors failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  res = hipGraphicsResourceGetMappedPointer
      ((void **)&cellsBufferData, &num_bytes, vboResources[3]);
  if(res != hipSuccess)
  {
    std::string errormsg = hipGetErrorString(res);
    cerr << "Get mappedpointer for cell indices failed ... "
         << hipGetErrorString(res) << endl;
    return;
  }

  // Copy on card verts to the shared on card gl buffer
  thrust::copy(pD->points->begin(), pD->points->end(),
               thrust::device_ptr<float3>(vertexBufferData));

  // Copy on card cell indices to the shared on card gl buffer
  if (pD->cells) {
    useindexbuffers = true;
    thrust::copy(pD->cells->begin(), pD->cells->end(),
                 thrust::device_ptr<uint3>(cellsBufferData));
  }

  hasNormals = false;
  if (pD->normals)
    {
    hasNormals = true;

    // Copy on card verts to the shared on card gl buffer
    thrust::copy(pD->normals->begin(), pD->normals->end(),
                 thrust::device_ptr<float>(normalsBufferData));
    }
  hasColors = false;


  if (pD->colors)
  {
//    thrust::fill(pD->colors->begin(), pD->colors->end(), 127);
    thrust::copy(pD->colors->begin(), pD->colors->end(), 
      thrust::device_ptr<float4>(colorbufferdata));
  }
//  else 
/*
  if (pD->scalars)
    {
    double scalarRange[2];
    id->GetScalarsRange(scalarRange);

    hasColors = true;

//    if(id->GetMTime() > dataObjectMTimeCache)
      {
      vtkPiston::minmax_pair<float> result = vtkPiston::find_min_max(
                                              pD->scalars);

      scalarRange[0] = static_cast<double>(result.min_val);
      scalarRange[1] = static_cast<double>(result.max_val);

      // Set parameters to compute scalars colors
      const int numvalues = 256;
      id->SetScalarsRange(scalarRange);
      psc->SetTableRange(scalarRange[0], scalarRange[1]);
      psc->SetNumberOfValues(numvalues);
      }

    std::vector<float> *colors = psc->ComputeScalarsColorsf(VTK_RGB);

    // Copy to GPU
    thrust::device_vector<float> onGPU(colors->begin(), colors->end());
    float *raw_ptr = thrust::raw_pointer_cast(&onGPU[0]);

    // Now run each scalar data through the map to choose a color for it

    // \NOTE: Since GPU most likely going to calculate range using single
    // floating point precision, we may lose precision and hence, we need
    // to check if the range min and max are almost equal
    //TODO: Remove this when piston gives us exactly same values for
    //isocontour.
    float tempRange[2] =
      {
      static_cast<float>(scalarRange[0]),
      static_cast<float>(scalarRange[1])
      };
    if( AlmostEqualRelativeAndAbs(scalarRange[0], scalarRange[1],
                                  numeric_limits<float>::epsilon(),
                                  numeric_limits<float>::epsilon() * 10) )
      {
      tempRange[1] = tempRange[0]+1.0;
      }

    color_map<float> colorMap(raw_ptr, onGPU.size(), VTK_RGB, tempRange[0], tempRange[1]);
    thrust::copy(thrust::make_transform_iterator(pD->scalars->begin(), colorMap),
                 thrust::make_transform_iterator(pD->scalars->end(), colorMap),
                 thrust::device_ptr<float3>(colorsBufferData));
    }
*/
  // Allow GL to access again
  res = hipGraphicsUnmapResources(4, vboResources, 0);
  if (res != hipSuccess)
  {
    cerr << "Release from CUDA failed ... " << hipGetErrorString(res) << endl;
    return;
  }

  return;
}
//------------------------------------------------------------------------------
} //namespace
